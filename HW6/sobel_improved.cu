
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include "string.h"


#define DEFAULT_THRESHOLD  4000

#define DEFAULT_FILENAME "BWstop-sign.ppm"

unsigned int *read_ppm( char *filename, int * xsize, int * ysize, int *maxval ){
  
  if ( !filename || filename[0] == '\0') {
    fprintf(stderr, "read_ppm but no file name\n");
    return NULL;  // fail
  }

  FILE *fp;

  fprintf(stderr, "read_ppm( %s )\n", filename);
  fp = fopen( filename, "rb");
  if (!fp) {
      fprintf(stderr, "read_ppm()    ERROR  file '%s' cannot be opened for reading\n", filename);
      return NULL; // fail 

  }

  char chars[1024];
  //int num = read(fd, chars, 1000);
  int num = fread(chars, sizeof(char), 1000, fp);

  if (chars[0] != 'P' || chars[1] != '6') 
    {
      fprintf(stderr, "Texture::Texture()    ERROR  file '%s' does not start with \"P6\"  I am expecting a binary PPM file\n", filename);
      return NULL;
    }

  unsigned int width, height, maxvalue;


  char *ptr = chars+3; // P 6 newline
  if (*ptr == '#') // comment line! 
    {
      ptr = 1 + strstr(ptr, "\n");
    }

  num = sscanf(ptr, "%d\n%d\n%d",  &width, &height, &maxvalue);
  fprintf(stderr, "read %d things   width %d  height %d  maxval %d\n", num, width, height, maxvalue);  
  *xsize = width;
  *ysize = height;
  *maxval = maxvalue;
  
  unsigned int *pic = (unsigned int *)malloc( width * height * sizeof(unsigned int));
  if (!pic) {
    fprintf(stderr, "read_ppm()  unable to allocate %d x %d unsigned ints for the picture\n", width, height);
    return NULL; // fail but return
  }

  // allocate buffer to read the rest of the file into
  int bufsize =  3 * width * height * sizeof(unsigned char);
  if ((*maxval) > 255) bufsize *= 2;
  unsigned char *buf = (unsigned char *)malloc( bufsize );
  if (!buf) {
    fprintf(stderr, "read_ppm()  unable to allocate %d bytes of read buffer\n", bufsize);
    return NULL; // fail but return
  }





  // TODO really read
  char duh[80];
  char *line = chars;

  // find the start of the pixel data.   no doubt stupid
  sprintf(duh, "%d\0", *xsize);
  line = strstr(line, duh);
  //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
  line += strlen(duh) + 1;

  sprintf(duh, "%d\0", *ysize);
  line = strstr(line, duh);
  //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
  line += strlen(duh) + 1;

  sprintf(duh, "%d\0", *maxval);
  line = strstr(line, duh);


  fprintf(stderr, "%s found at offset %d\n", duh, line - chars);
  line += strlen(duh) + 1;

  long offset = line - chars;
  //lseek(fd, offset, SEEK_SET); // move to the correct offset
  fseek(fp, offset, SEEK_SET); // move to the correct offset
  //long numread = read(fd, buf, bufsize);
  long numread = fread(buf, sizeof(char), bufsize, fp);
  fprintf(stderr, "Texture %s   read %ld of %ld bytes\n", filename, numread, bufsize); 

  fclose(fp);


  int pixels = (*xsize) * (*ysize);
  for (int i=0; i<pixels; i++) pic[i] = (int) buf[3*i];  // red channel

 

  return pic; // success
}


void write_ppm( char *filename, int xsize, int ysize, int maxval, int *pic) 
{
  FILE *fp;
  
  fp = fopen(filename, "w");
  if (!fp) 
    {
      fprintf(stderr, "FAILED TO OPEN FILE '%s' for writing\n");
      exit(-1); 
    }
  
  
  
  fprintf(fp, "P6\n"); 
  fprintf(fp,"%d %d\n%d\n", xsize, ysize, maxval);
  
  int numpix = xsize * ysize;
  for (int i=0; i<numpix; i++) {
    unsigned char uc = (unsigned char) pic[i];
    fprintf(fp, "%c%c%c", uc, uc, uc); 
  }
  fclose(fp);

}

__global__ void sobel_kernel(int xsize, int ysize, unsigned int *pic, int *result, int thresh) {
  __shared__ unsigned int pic_d[100][100];
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
 //pic_d[i*xsize+j] = pic[i*xsize+j];
  // __syncthreads();
  if( i > 0 && i < ysize - 1 && j > 0 && j < xsize - 1) {
      int offset = i*xsize + j;
      int sum1 =  pic[ xsize * (i-1) + j+1 ] -     pic[ xsize*(i-1) + j-1 ] 
        + 2 * pic[ xsize * (i)   + j+1 ] - 2 * pic[ xsize*(i)   + j-1 ]
        +     pic[ xsize * (i+1) + j+1 ] -     pic[ xsize*(i+1) + j-1 ];
      
      int sum2 = pic[ xsize * (i-1) + j-1 ] + 2 * pic[ xsize * (i-1) + j ]  + pic[ xsize * (i-1) + j+1 ] - pic[xsize * (i+1) + j-1 ] - 2 * pic[ xsize * (i+1) + j ] - pic[ xsize * (i+1) + j+1 ];
      
      int magnitude =  sum1*sum1 + sum2*sum2;
      result[offset] = (magnitude > thresh) * 255;
  }
} 

int main( int argc, char **argv )
{

  int thresh = DEFAULT_THRESHOLD;
  char *filename;
  filename = strdup( DEFAULT_FILENAME);
  
  if (argc > 1) {
    if (argc == 3)  { // filename AND threshold
      filename = strdup( argv[1]);
       thresh = atoi( argv[2] );
    }
    if (argc == 2) { // default file but specified threshhold
      
      thresh = atoi( argv[1] );
    }

    fprintf(stderr, "file %s    threshold %d\n", filename, thresh); 
  }


  int xsize, ysize, maxval;
  unsigned int *pic = read_ppm( filename, &xsize, &ysize, &maxval ); 


  int numbytes =  xsize * ysize * 3 * sizeof( int );
  int result_size = numbytes;
//  int result_size = xsize * ysize * sizeof(int);
  int pic_size = xsize * ysize * sizeof(unsigned int);
  int *result = (int *) malloc( numbytes );
  if (!result) { 
    fprintf(stderr, "sobel() unable to malloc %d bytes\n", numbytes);
    exit(-1); // fail
  }

  int *out = result;

  for (int col=0; col<ysize; col++) {
    for (int row=0; row<xsize; row++) { 
      *out++ = 0; 
    }
  }

  int *result_d;
  unsigned int *pic_d;
  int blockX = 16;
  int blockY = 16; 
  hipMalloc((void **) &result_d, result_size);
  hipMalloc((void **) &pic_d, pic_size);
  hipMemcpy(pic_d, pic, pic_size, hipMemcpyHostToDevice);
  dim3 block(blockX, blockY);
  dim3 grid((xsize + blockX - 1)/blockX, (ysize + blockY - 1)/blockY);
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  hipEventRecord(start, 0);
  sobel_kernel<<<grid, block>>>(xsize, ysize, pic_d, result_d, thresh);
  hipError_t error = hipGetLastError();  
  if (hipSuccess != error)
    printf( "Error! %s\n",hipGetErrorString(error) );
  hipEventRecord(end, 0);

  hipEventSynchronize(end);
  float time;
  hipEventElapsedTime(&time, start, end);
  hipMemcpy(result, result_d, result_size, hipMemcpyDeviceToHost); 
  hipFree(result_d);
  hipFree(pic_d);

  printf("time: %fms\n", time);
  write_ppm( "result.ppm", xsize, ysize, 255, result);

  fprintf(stderr, "sobel done\n"); 

}

