#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#include "util.h" // to check that output is correct
using namespace std;
// const int N = 16;
// const int blocksize = 16;

__global__ void add(size_t vect_len, float * vect1, float *vect2, float *result){
    size_t const tid = threadIdx.x;
    if (tid >= vect_len)
        return;
    result[tid] = vect1[tid] + vect2[tid]; 
}

int main(int argc,char ** argv){
    hipEvent_t start=0;
    hipEvent_t stop=0;
    float time =0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    timeval t1, t2, t3, t4;
    size_t vect_len = parse_args(argc,argv);
    if(vect_len <= 0){
        return -1; }
    printf("Elements: %d\n",vect_len);
    /*if (vect_len > 100){
        cout << "this program has very naive thread layout, "
        <<"please use vector length of less than 100" << endl;
    }*/
    float vect1[vect_len];
    float vect2[vect_len];
    float result[vect_len];
    init_vects(vect_len,vect1,vect2);
    //add vectors together
    float *vect1_d;
    float *vect2_d;
    float *result_d;    
    hipMalloc((void**)&vect1_d,vect_len*sizeof(float));
    hipMalloc((void**)&vect2_d,vect_len*sizeof(float));
    hipMalloc((void**)&result_d,vect_len*sizeof(float));
    gettimeofday(&t1, NULL);
    hipMemcpy(vect1_d,vect1,vect_len*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(vect2_d,vect2,vect_len*sizeof(float),hipMemcpyHostToDevice);
    //setup block and grid size
    dim3 dimBlock(vect_len,1);
    dim3 dimGrid(1,1);
    // call device kernel
    hipEventRecord(start,0);
    add<<<dimGrid,dimBlock>>>(vect_len,vect1_d,vect2_d,result_d);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time,start,stop);
    // copy data from device 
    hipMemcpy(result,result_d,vect_len*sizeof(float),hipMemcpyDeviceToHost);
    gettimeofday(&t2, NULL);
    // free device memory    
    hipFree(vect1_d);
    hipFree(vect2_d);
    hipFree(result_d);
    double  elapsedTime = (t2.tv_sec - t1.tv_sec)*1000.0 + (t2.tv_usec - t1.tv_usec)/1000.0;
    printf("\tGPU time (ms): %.4f\n\t\tKernel Time: %.4f\n",elapsedTime,time);
    gettimeofday(&t3, NULL);
    bool correct = verify(vect_len,vect1,vect2,result);
    gettimeofday(&t4, NULL);
    elapsedTime = (t4.tv_sec - t3.tv_sec)*1000.0 + (t4.tv_usec - t3.tv_usec)/1000.0;
    printf("\tCPU time (ms): %.4f\n",elapsedTime);
    if(correct){
    //    cout << "result is correct" << endl;
    } else{
        cout << "result is _NOT_ correct" << endl;
    }
    return 0;
}
